#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void kernelA() {
    // TEORIA : https://anuradha-15.medium.com/cuda-thread-indexing-fb9910cba084
    //blockDim.x — number of threads in the x dimension if the grid (eg:4)
    //blockIdx.x — block’s index in x dimension
    //ThreadIdx.x — thread’s index in x dimension

        //1D grid, 1d BLOCK
        //threadId = (blockIdx.x * blockDim.x) + threadIdx.x
        //Let’s check the equation for Thread(2, 0) in Block(1, 0).
        //Thread ID = (1 * 3) + 2 = 3 + 2 = 5

    int globalThreadId = (blockIdx.x * blockDim.x) + threadIdx.x;

    printf("My threadIdx.x is %d, blockIdx.x is %d, blockDim.x is %d, Global thread id is %d\n",
        threadIdx.x, blockIdx.x, blockDim.x, globalThreadId);

    //1D grid of 2D blocks
    //threadId = (blockIdx.x * blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x

    //2D grid of 1D blocks
    //threadId = (blockId * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x
}

int main()
{
 
    hipSetDevice(0);
    kernelA << <4, 3 >> > ();
    hipDeviceSynchronize();
    hipDeviceReset();

    return 0;
}