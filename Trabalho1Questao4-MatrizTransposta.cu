#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>


#include <random>
#include <time.h>

#define N 5

__global__ void transposta(float* M, float* T) {


	int C = blockDim.x * blockIdx.x + threadIdx.x;
	int L = blockDim.y * blockIdx.y + threadIdx.y;

	if (C < N && L < N) {
		T[C * N + L] = M[C + L * N];
	}
}



int main(void) {



	float* M_h;
	float* T_h;
	float* M_d;
	float* T_d;

	size_t size = N * N * sizeof(float);

	hipHostMalloc((float**)&M_h, size);
	T_h = (float*)malloc(size);
	hipMalloc((float**)&M_d, size);


	// init matrix
	for (int i = 0; i < N * N; ++i) {
		M_h[i] = rand() % 10;
	}

	hipMemcpyAsync(M_d, M_h, size, hipMemcpyHostToDevice);
	hipMalloc((float**)&T_d, size);
	hipMemset(T_d, 0, size);

	printf("\nMATRIZ GERADA\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			int num = M_h[i * N + j];
			printf(" %d ", num);
		}
		printf("\n");
	}





	int blocksizeX = N;
	int blocksizeY = N;
	int A = (N + blocksizeX - 1) / blocksizeX;
	int B = (N + blocksizeY - 1) / blocksizeY;

	dim3 block(blocksizeX, blocksizeY);
	dim3 grid(A, B);



	transposta << <grid, block >> > (M_d, T_d);

	hipMemcpy(T_h, T_d, size, hipMemcpyDeviceToHost);

	printf("\nTransposta\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			int num = T_h[i * N + j];
			printf(" %d ", num);
		}
		printf("\n");
	}




	hipFree(T_d);
	hipFree(M_d);
	return 0;
}