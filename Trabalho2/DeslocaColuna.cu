#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <random>
#include <time.h>
#include <math.h>


#define BlockSize 32

//N é o tamanho da MATRIZ
#define N 8

__global__ void deslocaColuna(float* M, float* T) {

	//Coluna
	int C = blockDim.x * blockIdx.x + threadIdx.x;
	//Linha
	int L = blockDim.y * blockIdx.y + threadIdx.y;

	if (C < N && L < N) {

		if (C >= 0 ) { //Desloca as colunas da matriz
			T[C + L * N + 1] = M[C + L * N];
		}
		if (C == N - 1) { //Bordas da Matriz
			printf("\n BORDAS ## Row/Col = (%d,%d)", L, C);
			//transforma o primeiro elemento da linha para ser o mesmo elementa dessa linha porém na ultima coluna
			T[L * N] = M[C + L * N];
		}
		
		
	}
}



int main(void) {

	float* M_h;
	float* T_h;
	float* M_d;
	float* T_d;

	float time;
	hipEvent_t start, stop;

	size_t size = N * N * sizeof(float);

	hipHostMalloc((float**)&M_h, size);
	T_h = (float*)malloc(size);
	hipMalloc((float**)&M_d, size);

	// init matrix
	for (int i = 0; i < N * N; ++i) {
		M_h[i] = rand() % 10;
	}

	hipMemcpyAsync(M_d, M_h, size, hipMemcpyHostToDevice);
	hipMalloc((float**)&T_d, size);
	hipMemset(T_d, 0, size);

	printf("\nMATRIZ GERADA\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			int num = M_h[i * N + j];
			printf(" %d ", num);
		}
		printf("\n");
	}


	//two dimension threads
	dim3 dimBlock(BlockSize, BlockSize);
	dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x, (N + dimBlock.y - 1) / dimBlock.y);


	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	deslocaColuna << <dimGrid, dimBlock >> > (M_d, T_d);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipMemcpy(T_h, T_d, size, hipMemcpyDeviceToHost);
	printf("\n\ndeslocaColuna\n");
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; ++j) {
			int num = T_h[i * N + j];
			printf(" %d ", num);
		}
		printf("\n");
	}



	printf("\n Tempo para gerar a matriz:  %3.5f ms \n", time);


	hipFree(T_d);
	hipFree(M_d);
	return 0;
}