#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdio.h>
#include <random>
#include <time.h>
#include <math.h>
__global__ void grids()
{
    int  threadRowID, threadColId;

    threadRowID = blockIdx.x * blockDim.x + threadIdx.x;
    threadColId = blockIdx.y * blockDim.y + threadIdx.y;

    /* ------------------------------------
       Print the thread's 2 dim grid ID
       ------------------------------------ */
    printf("Block: (%d,%d) Thread: (%d,%d) -> Row/Col = (%d,%d)\n",
        blockIdx.x, blockIdx.y,
        threadIdx.x, threadIdx.y,
        threadRowID, threadColId);
}

int main()
{
    dim3 blockShape = dim3(2, 2);
    dim3 gridShape = dim3(2, 2);

    grids << < gridShape, blockShape >> > ();  // Launch a 2 dim grid of threads  

  

    hipDeviceSynchronize();

    return 0;
}