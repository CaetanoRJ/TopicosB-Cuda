#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

//bibliotecas para gerar numeros aleatorios
#include <random>
#include <time.h>

#define arraySize 10

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	
	int a[arraySize] = {};
	int b[arraySize] = {};
	int c[arraySize] = {};

	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	srand(time(0));
	//populate the arrays A and B
	for (int i = 0; i < arraySize; i++) {
		
			a[i] = rand() % 10;
			b[i] = rand() % 10;
		}
	
	/* VETOR a */
	printf("VETOR A\n");
	for (int i = 0; i < arraySize; i++) {
		printf("%d\t", a[i]);
	}

	/* VETOR B */
	printf("\nVETOR B\n");
	for (int i = 0; i < arraySize; i++) {
		printf("%d\t", b[i]);
	}

	// Alocar espaço na memória do device
	cudaStatus = hipMalloc((void**)&dev_c, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copia os vetores do host para a device
	cudaStatus = hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}



	// Executar o kernel
	addKernel << <arraySize, 1 >> > (dev_c, dev_a, dev_b);

	// Verificar se o kernel foi executado corretamente
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Espera o kernel terminar e retorna quaisquer erros encontrados durante a execução
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copia o resultado do device para a memória do host.
	cudaStatus = hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	/* VETOR C */
	printf("\nVETOR C\n");
	for (int i = 0; i < arraySize; i++) {
		printf("%d\t", c[i]);
	}

	// Limpa a memória
Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}